#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include <stdio.h>
#include <iostream>
#include <typeinfo>
#include <list>
#include <cmath>
#include <chrono>
#include <time.h> 

using namespace std;



#pragma region ������� ����
const int THREADS_PER_BLOCK = 1024;

// ������� ���� ��������� ������
__global__ void multMatrixesKernel(double* A, double* B, double* C, int columnsA, int columnsB)
{
    int i0 = columnsA * (blockDim.y * blockIdx.y + threadIdx.y);
    int j0 = blockDim.x * blockIdx.x + threadIdx.x;
    double sum = 0;
    for (int k = 0; k < columnsA; k++)
        sum += A[i0 + k] * B[k * columnsB + j0];
    
    int index = columnsB * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    C[index] = sum;
}

// ������� ���� ��������� ������� �� �����
__global__ void multMatrixKernel(double* matrix, unsigned int size, double value) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        matrix[index] = matrix[index] * value;
}

// ������� ���� ���������������� �������
__global__ void transporseMatrixKernel(double* result, double* matrix, unsigned int rows, unsigned int columns) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // ����� ������ �� X
    int y = blockIdx.y * blockDim.y + threadIdx.y; // ����� ������ �� Y
    // (x, y) - ����������, ������� ��������� �������������� ����� � �����

    int i = y * columns + x; // ������ �������� � �������� �������
    int j = x * rows + y; // ������ �������� � ����������������� �������

    int count = rows * columns;
    if (i < count && j < count) 
    {
        result[j] = matrix[i];
    }
}

__device__ unsigned long long int atomicCAS(unsigned long long int* address, unsigned long long int compare, unsigned long long int val);
__device__ long long int __double_as_longlong(double x);
__device__ double __longlong_as_double(long long int x);
// ��������� �������� ��������� �����
__device__ double atomicMult(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val * __longlong_as_double(assumed)));

    } while (assumed != old);

    return __longlong_as_double(old);
}
__device__ void __syncthreads();

// ������� ���� ���������� ������������
__global__ void determinantMatrixKernel(double* triangular, unsigned int size, double* det) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // ����� ������ �� X
    int y = blockIdx.y * blockDim.y + threadIdx.y; // ����� ������ �� Y

    if (y == x && y < size) 
    {
        int index = y * size + y;
        double value = triangular[index];
        atomicMult(det, value);
    }
}

__global__ void reductionDeterminantMatrixKernel_1(double* triangular, unsigned int size, double* resultsBlocks)
{
    __shared__ double blockData[THREADS_PER_BLOCK];

    int x = blockIdx.x * blockDim.x + threadIdx.x; // ����� ������ �� X
    int y = blockIdx.y * blockDim.y + threadIdx.y; // ����� ������ �� Y

    if (y == x && y < size)
    {
        int threadId = threadIdx.x;
        int triangularIndex = y * size + y;

        blockData[threadId] = triangular[triangularIndex];
        __syncthreads();

        for (int i = 1; i < THREADS_PER_BLOCK; i *= 2)
        {
            int index = 2 * i * threadId;
            if ((index + i) < THREADS_PER_BLOCK && blockData[index + i] != 0)
            {
                blockData[index] *= blockData[index + i];
                __syncthreads();
            }
        }
        if (threadId == 0)
            resultsBlocks[blockIdx.x] = blockData[0];
    }
}

__global__ void reductionDeterminantMatrixKernel_2(double* triangular, unsigned int size, double* resultsBlocks)
{
    __shared__ double blockData[THREADS_PER_BLOCK];

    int x = blockIdx.x * blockDim.x + threadIdx.x; // ����� ������ �� X
    int y = blockIdx.y * blockDim.y + threadIdx.y; // ����� ������ �� Y

    if (y == x && y < size)
    {
        int threadId = threadIdx.x;
        int triangularIndex = y * size + y;

        blockData[threadId] = triangular[triangularIndex];
        __syncthreads();

        for (int i = 1; i < THREADS_PER_BLOCK; i *= 2)
        {
            int index = threadId + i;
            if (threadId % (2 * i) == 0 && blockData[index] != 0)
            {
                blockData[threadId] *= blockData[index];
                __syncthreads();
            }
        }
        if (threadId == 0)
            resultsBlocks[blockIdx.x] = blockData[0];
    }
}

__global__ void reductionDeterminantMatrixKernel_3(double* triangular, unsigned int size, double* resultsBlocks)
{
    __shared__ double blockData[THREADS_PER_BLOCK];

    int x = blockIdx.x * blockDim.x + threadIdx.x; // ����� ������ �� X
    int y = blockIdx.y * blockDim.y + threadIdx.y; // ����� ������ �� Y

    if (y == x && y < size)
    {
        int threadId = threadIdx.x;
        int triangularIndex = y * size + y;

        blockData[threadId] = triangular[triangularIndex];
        __syncthreads();

        for (int i = THREADS_PER_BLOCK / 2; i > 0; i >>= 1)
        {
            int index = threadId + i;
            if (threadId < i && blockData[index] != 0)
            {
                blockData[threadId] *= blockData[index];
            }
            __syncthreads();
        }
        if (threadId == 0)
            resultsBlocks[blockIdx.x] = blockData[0];
    }
}

__device__ void warpReduce(volatile double* sdata, int tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}
__global__ void reductionDeterminantMatrixKernel_5(double* triangular, unsigned int size, double* resultsBlocks)
{
    __shared__ double blockData[THREADS_PER_BLOCK];

    int x = blockIdx.x * blockDim.x + threadIdx.x; // ����� ������ �� X
    int y = blockIdx.y * blockDim.y + threadIdx.y; // ����� ������ �� Y

    if (y == x && y < size)
    {
        int threadId = threadIdx.x;
        int triangularIndex = y * size + y;

        blockData[threadId] = triangular[triangularIndex];
        __syncthreads();

        for (int i = THREADS_PER_BLOCK / 2; i > 0; i >>= 1)
        {
            int index = threadId + i;
            if (threadId < i&& blockData[index] != 0)
            {
                blockData[threadId] *= blockData[index];
            }
            __syncthreads();
        }

        for (unsigned int i = THREADS_PER_BLOCK / 2; i > 32; i >>= 1)
        {
            int index = threadId + i;
            if (threadId < i && blockData[index] != 0)
                blockData[threadId] *= blockData[index];
            __syncthreads();
        }
        if (threadId < 32)
            warpReduce(blockData, threadId);

        if (threadId == 0)
            resultsBlocks[blockIdx.x] = blockData[0];
    }
}

template <unsigned int blockSize>
__device__ void warpReduceTemp(volatile double* sdata, int tid)
{
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
template <unsigned int blockSize>
__global__ void reductionDeterminantMatrixKernel_6(double* triangular, unsigned int size, double* resultsBlocks)
{
    __shared__ double blockData[THREADS_PER_BLOCK];

    int x = blockIdx.x * blockDim.x + threadIdx.x; // ����� ������ �� X
    int y = blockIdx.y * blockDim.y + threadIdx.y; // ����� ������ �� Y

    if (y == x && y < size)
    {
        int threadId = threadIdx.x;
        int triangularIndex = y * size + y;

        blockData[threadId] = triangular[triangularIndex];
        __syncthreads();

        for (int i = THREADS_PER_BLOCK / 2; i > 0; i >>= 1)
        {
            int index = threadId + i;
            if (threadId < i&& blockData[index] != 0)
            {
                blockData[threadId] *= blockData[index];
            }
            __syncthreads();
        }

        if (THREADS_PER_BLOCK >= 512) {
            if (threadId < 256) { blockData[threadId] += blockData[threadId + 256]; } __syncthreads();
        }
        if (THREADS_PER_BLOCK >= 256) {
            if (threadId < 128) { blockData[threadId] += blockData[threadId + 128]; } __syncthreads();
        }
        if (THREADS_PER_BLOCK >= 128) {
            if (threadId < 64) { blockData[threadId] += blockData[threadId + 64]; } __syncthreads();
        }
        if (threadId < 32) warpReduceTemp<THREADS_PER_BLOCK>(blockData, threadId);

        if (threadId == 0)
            resultsBlocks[blockIdx.x] = blockData[0];
    }
}


#pragma endregion

void displayDeviceInfo();
void multMatrixesWithCuda(double* C, double* A, double* B, unsigned int rowsA, unsigned int columnsA, unsigned int rowsB, unsigned int columnsB);
void multMatrixWithCuda(double* matrix, unsigned int rows, unsigned int columns, double value);
void transporseMatrixWithCuda(double* result, double* matrix, unsigned int rows, unsigned int columns);
double getDeterminantWithCuda(double* matrix, unsigned int rows, unsigned int columns);
double getDeterminantWithCudaReduction(double* matrix, unsigned int rows, unsigned int columns, int reductionNum, double* elapsedKernel);
double* getTriangularMatrix(double* matrix, unsigned int rows, unsigned int columns);
double* generateMatrix(int rows, int columns, int minValue, int maxValue);

int main()
{
    int countRepeats = 1;
    double time;
    auto start = chrono::high_resolution_clock::now();
    auto end = chrono::high_resolution_clock::now();
    chrono::duration<double, std::milli> elapsed = end - start;

    unsigned int rowsMatrixA, columnsMatrixA;
    rowsMatrixA = 1000;
    columnsMatrixA = 1000;
    double* matrixA = generateMatrix(rowsMatrixA, columnsMatrixA, 1, 99);

    unsigned int rowsMatrixB, columnsMatrixB;
    rowsMatrixB = 1000;
    columnsMatrixB = 1000;
    double* matrixB = generateMatrix(rowsMatrixB, columnsMatrixB, 1, 99);

    // �������� ����� ������ ��� ������������� ����� ���������� CUDA
    hipSetDevice(0);
    getDeterminantWithCuda(matrixA, rowsMatrixA, columnsMatrixA);
    cout << "===================" << "\n\n";

#pragma region TestTransporseMatrix
    double* resultTransporse = new double[rowsMatrixA * columnsMatrixA];

    start = chrono::high_resolution_clock::now();
    for (size_t i = 0; i < countRepeats; i++)
    {
        transporseMatrixWithCuda(resultTransporse, matrixA, rowsMatrixA, columnsMatrixA);
    }
    end = chrono::high_resolution_clock::now();

    elapsed = end - start;
    time = elapsed.count() / countRepeats;
    //cout << "Time transpose method (ms): " << time << "\n\n";
    
    delete[] resultTransporse;
#pragma endregion

#pragma region TestMultMatrixes
    double* resultMultMatrixes = new double[columnsMatrixA * rowsMatrixB];

    start = chrono::high_resolution_clock::now();
    for (size_t i = 0; i < countRepeats; i++)
    {
        multMatrixesWithCuda(resultMultMatrixes, matrixA, matrixB, rowsMatrixA, columnsMatrixA, rowsMatrixB, columnsMatrixB);
    }
    end = chrono::high_resolution_clock::now();

    elapsed = end - start;
    time = elapsed.count() / countRepeats;
    //cout << "Time multiplication matrix method (ms): " << time << "\n\n";
    
    delete[] resultMultMatrixes;
#pragma endregion

#pragma region TestDeterminantMatrix
    start = chrono::high_resolution_clock::now();
    for (size_t i = 0; i < countRepeats; i++)
    {
        double det = getDeterminantWithCuda(matrixA, rowsMatrixA, columnsMatrixA);
        //cout << "det = " << det << "\n";
    }
    end = chrono::high_resolution_clock::now();

    elapsed = end - start;
    time = elapsed.count() / countRepeats;
    //cout << "Time determinant matrix method (ms): " << time << "\n\n";
#pragma endregion

#pragma region TestReductionDeterminant
    for (int reductionNum = 1; reductionNum <= 6; reductionNum++)
    {
        if (reductionNum == 4) continue;
        double sumElapsedKernel = 0;
        double* elapsedKernel = (double*)malloc(sizeof(double));
        //start = chrono::high_resolution_clock::now();
        for (size_t i = 0; i < countRepeats; i++)
        {
            getDeterminantWithCudaReduction(matrixA, rowsMatrixA, columnsMatrixA, reductionNum, elapsedKernel);
            sumElapsedKernel += *elapsedKernel;
        }
        //end = chrono::high_resolution_clock::now();

        //elapsed = end - start;
        //time = elapsed.count() / countRepeats;
        time = sumElapsedKernel / countRepeats;
        cout << "Time reduction " << reductionNum << " determinant matrix kernel method(ms): " << time << "\n\n";
    }

    /*elapsed = end - start;
    time = elapsed.count() / countRepeats;
    cout << "Time reduction determinant matrix method (ms): " << time << "\n\n";*/
#pragma endregion

#pragma region TestMultMatrix
    start = chrono::high_resolution_clock::now();
    for (size_t i = 0; i < countRepeats; i++)
    {
        multMatrixWithCuda(matrixA, rowsMatrixA, columnsMatrixA, 2);
    }
    end = chrono::high_resolution_clock::now();

    elapsed = end - start;
    time = elapsed.count() / countRepeats;
    //cout << "Time multiplication matrix by number method (ms): " << time << "\n\n";
#pragma endregion

    return 0;
}

// ����� ���������� �� ���������� (GPU)
void displayDeviceInfo()
{
    const int kb = 1024;
    const int mb = kb * kb;
    std::wcout << "NBody.GPU" << std::endl << "=========" << std::endl << std::endl;

    wcout << "CUDA version:   v" << CUDART_VERSION << endl;
    //wcout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << endl << endl;

    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "CUDA Devices: " << endl << endl;

    for (int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

        wcout << "  Warp size:         " << props.warpSize << endl;
        wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }
}

// ���������������� �������
void transporseMatrixWithCuda(double* result, double* matrix, unsigned int rows, unsigned int columns)
{
    // ��������� ������ �� ����������
    double* devMatrix;
    double* devResult;
    hipMalloc((void**)&devMatrix, rows * columns * sizeof(double));
    hipMalloc((void**)&devResult, rows * columns * sizeof(double));

    // ����������� �������� � ������ ����� �� ����������
    hipMemcpy(devMatrix, matrix, rows * columns * sizeof(double), hipMemcpyHostToDevice);

    // ���������� �������� ����� � �����
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    int threadsPerBlockDim = sqrt(props.maxThreadsPerBlock); // Max threads for block = 1024 = 32*32
    dim3 blockDim(threadsPerBlockDim, threadsPerBlockDim); // 2D-block 32*32

    int blocksPerGridDimX = ceilf(columns / (float)threadsPerBlockDim);
    int blocksPerGridDimY = ceilf(rows / (float)threadsPerBlockDim);
    dim3 gridDim(blocksPerGridDimX, blocksPerGridDimY);

    auto start = chrono::high_resolution_clock::now();

    // ������ ������� ����
    transporseMatrixKernel<<< gridDim, blockDim >>>(devResult, devMatrix, rows, columns);
    hipDeviceSynchronize();

    auto end = chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    cout << "Time transporse matrix kernel method (ms): " << elapsed.count() << std::endl;

    hipMemcpy(result, devResult, rows * columns * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(devResult);
    hipFree(devMatrix);
}

// ��������� ������
void multMatrixesWithCuda(double* C, double* A, double* B, unsigned int rowsA, unsigned int columnsA, unsigned int rowsB, unsigned int columnsB)
{
    if (columnsA != rowsB)
        throw new std::exception("���������� ����� � �������� ������������� ������ �� ���������.");

    double *devC, *devA, *devB;

    // ��������� ������ �� ����������
    hipMalloc((void**)&devC, columnsA * rowsB * sizeof(double));
    hipMalloc((void**)&devA, columnsA * rowsA * sizeof(double));
    hipMalloc((void**)&devB, columnsB * rowsB * sizeof(double));

    // ����������� ������ � ����� �� ����������
    hipMemcpy(devA, A, columnsA * rowsA * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, columnsB * rowsB * sizeof(double), hipMemcpyHostToDevice);
    
    // ���������� �������� ����� � �����
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    int threadsPerBlockDim = sqrt(props.maxThreadsPerBlock);
    dim3 blockDim(threadsPerBlockDim, threadsPerBlockDim);

    int blocksPerGridDimX = ceilf(columnsA / threadsPerBlockDim);
    int blocksPerGridDimY = ceilf(rowsB / threadsPerBlockDim);
    dim3 gridDim(blocksPerGridDimX, blocksPerGridDimY);

    auto start = chrono::high_resolution_clock::now();
    
    // ������ ������� ����
    multMatrixesKernel << < gridDim, blockDim >> > (devA, devB, devC, columnsA, columnsB);
    hipDeviceSynchronize();

    auto end = chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    cout << "Time multiplication matrixes kernel method (ms): " << elapsed.count() << std::endl;

    // ����������� ���������� � ���������� �� ����
    hipMemcpy(C, devC, columnsA * rowsB * sizeof(double), hipMemcpyDeviceToHost);

    // ������������ ������ ����������
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

// ��������� ������� �� �����
void multMatrixWithCuda(double* matrix, unsigned int rows, unsigned int columns, double value) 
{
    int size = rows * columns;

    double* devMatrix;
    hipMalloc((void**)&devMatrix, size * sizeof(double));
    hipMemcpy(devMatrix, matrix, size * sizeof(double), hipMemcpyHostToDevice);

    // ���������� �������� ����� � �����
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    int threadsPerBlock = props.maxThreadsPerBlock;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    auto start = chrono::high_resolution_clock::now();

    // ������ ������� ����
    multMatrixKernel<<<blocksPerGrid, threadsPerBlock>>>(devMatrix, size, value);
    hipDeviceSynchronize();

    auto end = chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    cout << "Time multiplication matrix by number kernel method (ms): " << elapsed.count() << std::endl;

    // ����������� ���������� � ���������� �� ����
    hipMemcpy(matrix, devMatrix, size * sizeof(double), hipMemcpyDeviceToHost);

    // ������������ ������ ����������
    hipFree(devMatrix);
}

// ��������� ����������� �������
double* getTriangularMatrix(double* matrix, unsigned int rows, unsigned int columns) 
{
    if (rows != columns)
        throw std::invalid_argument("������� �� �������� ������������.");

    double* result = new double[rows * columns];
    for (size_t i = 0; i < rows; i++)
    {
        for (size_t j = 0; j < columns; j++)
        {
            result[i * columns + j] = matrix[i * columns + j];
        }
    }

    for (int i = 0; i < rows - 1; i++)
    {
        for (int j = i + 1; j < columns; j++)
        {
            double coef;
            if (result[i * columns + i] == 0)
            {
                for (int k = 0; k < columns; k++)
                {
                    result[i * columns + k] += result[(i + 1) * columns + k];
                }
                coef = result[j * columns + i] / result[i * columns + i];
            }
            else
            {
                coef = result[j * columns + i] / result[i * columns + i];
            }

            if (std::isnan(coef)) 
                coef = 0;

            for (int k = i; k < rows; k++)
            {
                result[j * rows + k] -= result[i * rows + k] * coef;
            }
        }
    }

    return result;
}

// ��������� ������������
double getDeterminantWithCuda(double* matrix, unsigned int rows, unsigned int columns)
{
    if (rows != columns) 
        throw std::invalid_argument("������� �� �������� ������������.");

    double* triangular = getTriangularMatrix(matrix, rows, columns);
    double* det = (double*)malloc(sizeof(double));
    *det = 1;

    // ��������� ������ �� ����������
    double* devTriangular;
    double* devDet;
    hipMalloc((void**)&devTriangular, columns * rows * sizeof(double));
    hipMalloc((void**)&devDet, sizeof(double));
    
    hipMemcpy(devTriangular, triangular, rows * columns * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devDet, det, sizeof(double), hipMemcpyHostToDevice);

    // ���������� �������� ����� � �����
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    int threadsPerBlockDim = sqrt(props.maxThreadsPerBlock);
    dim3 blockDim(threadsPerBlockDim, threadsPerBlockDim);

    int blocksPerGridDimX = ceilf(rows / (float)threadsPerBlockDim);
    int blocksPerGridDimY = ceilf(rows / (float)threadsPerBlockDim);
    dim3 gridDim(blocksPerGridDimX, blocksPerGridDimY);

    auto start = chrono::high_resolution_clock::now();

    // ������ ����
    determinantMatrixKernel<<<gridDim, blockDim>>> (devTriangular, rows, devDet);
    hipDeviceSynchronize();

    auto end = chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    cout << "Time determinant matrix kernel method (ms): " << elapsed.count() << std::endl;

    hipMemcpy(det, devDet, sizeof(double), hipMemcpyDeviceToHost);

    // ������������ ������ �� ����� � �� ����������
    double result = *det;

    hipFree(devTriangular);
    hipFree(devDet);
    
    delete[] triangular;
    free(det);
    
    return result;
}

// ��������� ������������ � ������� ��������
double getDeterminantWithCudaReduction(double* matrix, unsigned int rows, unsigned int columns, int reductionNum, double* elapsedKernel)
{
    if (rows != columns)
        throw std::invalid_argument("������� �� �������� ������������.");

    double* triangular = getTriangularMatrix(matrix, rows, columns);

    int size = rows;
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    int threadsPerBlock = props.maxThreadsPerBlock;
    //int blocks = ceilf((size + threadsPerBlock - 1) / (float)threadsPerBlock);

    int threadsPerBlockDim = sqrt(threadsPerBlock);
    dim3 blockDim(threadsPerBlockDim, threadsPerBlockDim);

    int blocksPerGridDim = ceilf(rows / (float)threadsPerBlockDim);
    int blocks = blocksPerGridDim * blocksPerGridDim;
    dim3 gridDim(blocksPerGridDim, blocksPerGridDim);

    // ��������� ������ �� ����������
    double* devTriangular;
    double* devResultsBlocks; // ���������� ���������� ������ �� ����������
    hipMalloc((void**)&devTriangular, rows * columns * sizeof(double));
    hipMalloc((void**)&devResultsBlocks, blocks * sizeof(double));
    hipMemcpy(devTriangular, triangular, rows * columns * sizeof(double), hipMemcpyHostToDevice);

    auto start = chrono::high_resolution_clock::now();
    // ������ ����
    switch (reductionNum)
    {
        case 1:
            reductionDeterminantMatrixKernel_1 << <gridDim, blockDim >> > (devTriangular, size, devResultsBlocks);

        case 2:
            reductionDeterminantMatrixKernel_2 << <gridDim, blockDim >> > (devTriangular, size, devResultsBlocks);
        case 3:
            reductionDeterminantMatrixKernel_3 << <gridDim, blockDim >> > (devTriangular, size, devResultsBlocks);
        case 5:
            reductionDeterminantMatrixKernel_5 << <gridDim, blockDim >> > (devTriangular, size, devResultsBlocks);
        case 6:
            reductionDeterminantMatrixKernel_6<THREADS_PER_BLOCK> << <gridDim, blockDim >> > (devTriangular, size, devResultsBlocks);

        default:
            reductionDeterminantMatrixKernel_1 << <gridDim, blockDim >> > (devTriangular, size, devResultsBlocks);
    }
    auto end = chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    *elapsedKernel = elapsed.count();

    hipDeviceSynchronize();

    double* resultsBlocks = (double*)malloc(blocks * sizeof(double)); // ���������� ���������� ������ �� �����
    hipMemcpy(resultsBlocks, devResultsBlocks, blocks * sizeof(double), hipMemcpyDeviceToHost);

    // ���������� ����������� ������
    double det = 1;
    for (size_t i = 0; i < blocks; i++)
    {
        if (resultsBlocks[i] != 0)
            det *= resultsBlocks[i];
    }

    // ������������ ������
    hipFree(devTriangular);
    hipFree(devResultsBlocks);
    delete[] triangular;
    free(resultsBlocks);

    return det;
}

// ��������� ������� ���������� �������
double* generateMatrix(int rows, int columns, int minValue, int maxValue) 
{
    double* matrix = new double[rows * columns];
    srand(time(NULL));
    double randValue;
    int precisionPoints = 2;

    for (size_t i = 0; i < rows; i++)
    {
        for (size_t j = 0; j < columns; j++)
        {
            randValue = rand() % (int)pow(10, precisionPoints);
            randValue = minValue + (randValue / pow(10, precisionPoints)) * (maxValue - minValue);
            matrix[i * columns + j] = randValue;
        }
    }

    return matrix;
}